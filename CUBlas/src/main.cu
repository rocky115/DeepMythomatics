/*

*Copyright (c) 2018 Radhamadhab Dalai
*Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated
documentation files (the "Software"), to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the Software, and
to permit persons to whom the Software is furnished to do so, subject to the following conditions: The above
copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO
THE WARRANTIES OF MERCHANTABILITY,FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.

 */


#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "my_cublas.h"

void perform_symv() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int n = 1024;
    float alpha = 1.0f;
    float beta = 0.0f;
    float *d_A, *d_x, *d_y;
    float *h_A, *h_x, *h_y;

    h_A = (float*)malloc(n * n * sizeof(float));
    h_x = (float*)malloc(n * sizeof(float));
    h_y = (float*)malloc(n * sizeof(float));

    hipMalloc((void**)&d_A, n * n * sizeof(float));
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    for (int i = 0; i < n * n; i++) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < n; i++) {
        h_x[i] = static_cast<float>(rand()) / RAND_MAX;
        h_y[i] = 0.0f;
    }

    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);

    hipblasSsymv(handle, HIPBLAS_FILL_MODE_LOWER, n, &alpha, d_A, n, d_x, 1, &beta, d_y, 1);

    hipMemcpy(h_y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result vector y (SYMV):\n";
    for (int i = 0; i < n; i++) {
        std::cout << h_y[i] << " ";
    }
    std::cout << "\n";

    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    free(h_A);
    free(h_x);
    free(h_y);
    hipblasDestroy(handle);
}

void perform_gemv() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int m = 1024, n = 1024;
    float alpha = 1.0f;
    float beta = 0.0f;
    float *d_A, *d_x, *d_y;
    float *h_A, *h_x, *h_y;

    h_A = (float*)malloc(m * n * sizeof(float));
    h_x = (float*)malloc(n * sizeof(float));
    h_y = (float*)malloc(m * sizeof(float));

    hipMalloc((void**)&d_A, m * n * sizeof(float));
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, m * sizeof(float));

    for (int i = 0; i < m * n; i++) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < n; i++) {
        h_x[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < m; i++) {
        h_y[i] = 0.0f;
    }

    hipMemcpy(d_A, h_A, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, m * sizeof(float), hipMemcpyHostToDevice);

    hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha, d_A, m, d_x, 1, &beta, d_y, 1);

    hipMemcpy(h_y, d_y, m * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result vector y (GEMV):\n";
    for (int i = 0; i < m; i++) {
        std::cout << h_y[i] << " ";
    }
    std::cout << "\n";

    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    free(h_A);
    free(h_x);
    free(h_y);
    hipblasDestroy(handle);
}

void perform_hemv() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int n = 1024;
    hipComplex alpha = make_hipComplex(1.0f, 0.0f);
    hipComplex beta = make_hipComplex(0.0f, 0.0f);
    hipComplex *d_A, *d_x, *d_y;
    hipComplex *h_A, *h_x, *h_y;

    h_A = (hipComplex*)malloc(n * n * sizeof(hipComplex));
    h_x = (hipComplex*)malloc(n * sizeof(hipComplex));
    h_y = (hipComplex*)malloc(n * sizeof(hipComplex));

    hipMalloc((void**)&d_A, n * n * sizeof(hipComplex));
    hipMalloc((void**)&d_x, n * sizeof(hipComplex));
    hipMalloc((void**)&d_y, n * sizeof(hipComplex));

    for (int i = 0; i < n * n; i++) {
        h_A[i] = make_hipComplex(static_cast<float>(rand()) / RAND_MAX, static_cast<float>(rand()) / RAND_MAX);
    }
    for (int i = 0; i < n; i++) {
        h_x[i] = make_hipComplex(static_cast<float>(rand()) / RAND_MAX, static_cast<float>(rand()) / RAND_MAX);
        h_y[i] = make_hipComplex(0.0f, 0.0f);
    }

    hipMemcpy(d_A, h_A, n * n * sizeof(hipComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, n * sizeof(hipComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(hipComplex), hipMemcpyHostToDevice);

    hipblasChemv(handle, HIPBLAS_FILL_MODE_LOWER, n, &alpha, d_A, n, d_x, 1, &beta, d_y, 1);

    hipMemcpy(h_y, d_y, n * sizeof(hipComplex), hipMemcpyDeviceToHost);

    std::cout << "Result vector y (HEMV):\n";
    for (int i = 0; i < n; i++) {
        std::cout << h_y[i].x << " + " << h_y[i].y << "i ";
    }
    std::cout << "\n";

    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    free(h_A);
    free(h_x);
    free(h_y);
    hipblasDestroy(handle);
}

int main() {
    perform_symv();
    perform_gemv();
    perform_hemv();
    return 0;
}
